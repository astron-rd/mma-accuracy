#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>

#include "gemm/gpu/GEMM_GPU.h"

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  } while (0)

GEMM_GPU::GEMM_GPU(size_t M, size_t N, size_t K) : GEMM(M, N, K) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));

  std::cout << "GPU: " << deviceProp.name << " (SM " << deviceProp.major << "."
            << deviceProp.minor << ")" << std::endl;

  CUDA_CHECK(hipMalloc(&d_a_, sizeof(float) * M * K));
  CUDA_CHECK(hipMalloc(&d_b_, sizeof(float) * K * N));
  CUDA_CHECK(hipMalloc(&d_c_, sizeof(float) * M * N));
}

GEMM_GPU::~GEMM_GPU() {
  CUDA_CHECK(hipFree(d_a_));
  CUDA_CHECK(hipFree(d_b_));
  CUDA_CHECK(hipFree(d_c_));
}

__global__ void kernel(const float *A, const float *B, float *C);

void GEMM_GPU::compute(const float *A, const float *B, float *C) {
  CUDA_CHECK(
      hipMemcpy(d_a_, A, sizeof(float) * M_ * K_, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(d_b_, B, sizeof(float) * K_ * N_, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(d_c_, C, sizeof(float) * M_ * N_, hipMemcpyHostToDevice));

  dim3 threads(32);
  dim3 grid(1);

  kernel<<<grid, threads>>>(d_a_, d_b_, d_c_);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(
      hipMemcpy(C, d_c_, sizeof(float) * M_ * N_, hipMemcpyDeviceToHost));
}
